// matrixVecMul.cu 
/* 
* Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.  
* 
* NOTICE TO USER:  
* 
* This source code is subject to NVIDIA ownership rights under U.S. and 
* international Copyright laws.  
* 
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.  
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
* OR PERFORMANCE OF THIS SOURCE CODE.  
* 
* U.S. Government End Users.  This source code is a "commercial item" as 
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
* "commercial computer software" and "commercial computer software 
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
* and is provided to the U.S. Government only as a commercial end item.  
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
* source code with only those rights set forth herein.  
*/ 
 
// Header files 

#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h> 
#include <string.h> 
 
// Types 
typedef struct { 
    int width; 
    int height; 
    int pitch; 
    float* e; 
} Matrix; 
 
typedef struct { 
    int width; 
    int height; 
    int pitch; 
    double* e; 
} MatrixD; 
 
typedef struct { 
    int length; 
    float* e; 
} Vector; 
 
typedef struct { 
    int length; 
    double* e; 
} VectorD; 
 
 
// Forward declarations 
void    MatrixVectorMulOnHost(const Matrix, const Vector, VectorD); 
void    MatrixVectorMulOnDevice(const Matrix, const Vector, Vector, int, int); 
 
Matrix  AllocateMatrix(int, int, float); 
void    FreeMatrix(Matrix); 
MatrixD AllocateMatrixD(int, int); 
void    FreeMatrixD(MatrixD); 
 
Matrix  AllocateDeviceMatrix(const Matrix); 
void    FreeDeviceMatrix(Matrix); 
void    CopyToDeviceMatrix(Matrix, const Matrix); 
void    CopyFromDeviceMatrix(Matrix, const Matrix); 
 
Vector  AllocateVector(int, float); 
void    FreeVector(Vector); 
VectorD AllocateVectorD(int); 
void    FreeVectorD(VectorD); 
 
Vector  AllocateDeviceVector(const Vector); 
void    FreeDeviceVector(Vector); 
void    CopyToDeviceVector(Vector, const Vector); 
void    CopyFromDeviceVector(Vector, const Vector); 
 
bool    CheckDevice(); 
double  ComputeL2NormError(const Vector, const VectorD); 
// The main function does the following: 
// - It initializes a matrix M and a vector V with random values 
// - It computes the product M * V on the device and on the host 
// - It compares the two results by computing the L2-norm error 
// The goal of the exercise is to replace TODO by the right piece of code 
// so that the source code compiles and produces an error of 0 
// to machine precision (less than 1e-6) 
int main(void){ 
    if (CheckDevice() == false) { 
        printf("\nThere is no device supporting CUDA.\n"); 
        printf("\nPress ENTER to exit...\n"); 
        getchar(); 
        return -1; 
    } 
 
    // 
    int maxThreadsPerBlock = 512; // need to optimize 
    int numMultiprocessors = 16; // change for GPU specs 
 
    // Need to modify the configuration 
    int numThreadsPerBlock = maxThreadsPerBlock; 
    int numBlocks = 2 * numMultiprocessors; 
    int width = 20 * numThreadsPerBlock; 
    int height = 20 * numBlocks;         
 
 
    // Allocate and initialize the matrices 
    Matrix  M  = AllocateMatrix(width, height, 1); 
    Vector  V  = AllocateVector(width, 1); 
    Vector  W  = AllocateVector(height, 0); 
    VectorD Wh = AllocateVectorD(height); 
 
    // M * V on the device 
    MatrixVectorMulOnDevice(M, V, W, numBlocks, numThreadsPerBlock); 
 
    // M * V on the host 
    MatrixVectorMulOnHost(M, V, Wh); 
 
    // Compute and print the L2-norm error 
    double error = ComputeL2NormError(W, Wh); 
    if (error < 1e-6) 
        printf("Correct result (L2-norm error is %e).\n", error); 
    else 
        printf("Incorrect result (L2-norm error is %e).\n", error); 
 
    printf("\nPress ENTER to exit...\n"); 
    getchar(); 
 
    // Free matrices 
    FreeMatrix(M); 
    FreeVector(V); 
    FreeVector(W); 
    FreeVectorD(Wh); 
 
    return 0; 
} 
 
// Forward declaration of the matrix vector multiplication kernel 
__global__ void MatrixVectorMulKernel(const Matrix, const Vector, Vector); 
 
// Matrix vector multiplication on the device 
void MatrixVectorMulOnDevice(const Matrix M, const Vector V, Vector W, 
                             int numBlocks, int numThreadsPerBlock) 
{ 
    // Load M and V to the device 
    Matrix Md = AllocateDeviceMatrix(M); 
    CopyToDeviceMatrix(Md, M); 
    Vector Vd = AllocateDeviceVector(V); 
    CopyToDeviceVector(Vd, V); 
 
    // Allocate W on the device 
    Vector Wd = AllocateDeviceVector(W); 
    CopyToDeviceVector(Wd, W); // Clear memory 
 
    // Launch the device computation 
    int sharedMemoryByteSize = numThreadsPerBlock * sizeof(float); 
    MatrixVectorMulKernel<<<numBlocks, numThreadsPerBlock, sharedMemoryByteSize>>>(Md, Vd, Wd); 
 
    // Read W from the device 
    CopyFromDeviceVector(W, Wd); 
 
    // Free device matrices 
    FreeDeviceMatrix(Md); 
    FreeDeviceVector(Vd); 
    FreeDeviceVector(Wd); 
} 
 
// Get a matrix row 
__device__ float* GetMatrixRow(const Matrix M, int y){ 
    return &M.e[y * M.pitch]; 
} 
 
// Get a vector element 
__device__ float GetVectorElement(const Vector V, int x){ 
    return V.e[x]; 
} 
 
// Set a vector element 
__device__ void SetVectorElement(Vector V, int x, float value){ 
    V.e[x] = value; 
} 
 
// Matrix vector multiplication kernel called by MatrixVectorMulOnDevice() 
__global__ void MatrixVectorMulKernel(const Matrix M, const Vector V, Vector W){ 
    // Number of blocks 
    int numBlocks = gridDim.x; 
 
    // Block ID 
    int blockID = blockIdx.x; 
 
    // Number of threads per block 
    int numThreadsPerBlock = blockDim.x; 
 
    // Thread ID 
    int threadID = threadIdx.x; 
 
    // Number of threads per grid 
    int numThreads = numBlocks * numThreadsPerBlock; 
       
    // Each thread handles as many matrix rows as necessary 
    for (int y = blockID * numThreadsPerBlock + threadID; 
         y < M.height; 
         y += numThreads) 
    { 
 
        // Pointer to the matrix row handled by thread threadID 
        const float* row = GetMatrixRow(M, y); 
 
        // Compute dot product 
        float dotProduct = 0; 
        for (int x = 0; x < M.width; ++x) 
            dotProduct += row[x] * GetVectorElement(V, x); 
 
        // Write result to global memory 
        SetVectorElement(W, y, dotProduct); 
    } 
} 
 
// Matrix vector multiplication on the host in double precision 
void MatrixVectorMulOnHost(const Matrix M, const Vector V, VectorD W){  
    for (int i = 0; i < M.height; ++i) { 
        double dotProduct = 0; 
        for (int k = 0; k < M.width; ++k) { 
            double a = M.e[i * M.width + k]; 
            double b = V.e[k]; 
            dotProduct += a * b; 
        } 
        W.e[i] = dotProduct; 
    } 
} 
 
// Allocate a matrix with random float entries 
Matrix AllocateMatrix(int width, int height, float scale){ 
    Matrix M; 
    M.width  = width; 
    M.height = height; 
    M.pitch  = width; 
    int size = width * height; 
    M.e = (float*)malloc(sizeof(float) * size); 
    for (int i = 0; i < size; ++i) 
        M.e[i] = scale * (rand() / (float)RAND_MAX); 
    return M; 
} 
 
// Free a matrix 
void FreeMatrix(Matrix M){ 
    free(M.e); 
} 
 
// Allocate a matrix with double entries 
MatrixD AllocateMatrixD(int width, int height){ 
    MatrixD M; 
    M.width  = width; 
    M.height = height; 
    M.pitch  = width; 
    M.e = (double*)malloc(sizeof(double) * width * height); 
    return M; 
} 
 
// Free a matrix with double entries 
void FreeMatrixD(MatrixD M){ 
    free(M.e); 
} 
 
// Allocate a device matrix of same size as M 
Matrix AllocateDeviceMatrix(const Matrix M){ 
    Matrix Mdevice = M; 
    int size = M.width * M.height * sizeof(float); 
    hipMalloc((void**)&Mdevice.e, size); 
    return Mdevice; 
} 
 
// Free a device matrix 
void FreeDeviceMatrix(Matrix M){ 
    hipFree(M.e); 
} 
 
// Copy a host matrix to a device matrix 
void CopyToDeviceMatrix(Matrix Mdevice, const Matrix Mhost){ 
    int size = Mhost.width * Mhost.height * sizeof(float); 
    hipMemcpy(Mdevice.e, Mhost.e, size, hipMemcpyHostToDevice); 
} 
 
// Copy a device matrix to a host matrix 
void CopyFromDeviceMatrix(Matrix Mhost, const Matrix Mdevice){ 
    int size = Mdevice.width * Mdevice.height * sizeof(float); 
    hipMemcpy(Mhost.e, Mdevice.e, size, hipMemcpyDeviceToHost); 
} 
 
// Allocate a vector with random float entries 
Vector AllocateVector(int length, float scale){ 
    Vector V; 
    V.length  = length; 
    V.e = (float*)malloc(sizeof(float) * length); 
    for (int i = 0; i < length; ++i) 
        V.e[i] = scale * (rand() / (float)RAND_MAX); 
    return V; 
} 
 
// Free a vector 
void FreeVector(Vector V){ 
    free(V.e); 
} 
 
// Allocate a vector with double entries 
VectorD AllocateVectorD(int length){ 
    VectorD V; 
    V.length  = length; 
    V.e = (double*)malloc(sizeof(double) * length); 
    return V; 
} 
 
// Free a vector with double entries 
void FreeVectorD(VectorD V){ 
    free(V.e); 
} 
 
// Allocate a device vector of same size as V 
Vector AllocateDeviceVector(const Vector V){ 
    Vector Vdevice = V; 
    int size = V.length * sizeof(float); 
    hipMalloc((void**)&Vdevice.e, size); 
    return Vdevice; 
} 
 
// Free a device vector 
void FreeDeviceVector(Vector V){ 
    hipFree(V.e); 
} 
 
// Copy a host vector to a device vector 
void CopyToDeviceVector(Vector Vdevice, const Vector Vhost){ 
    int size = Vhost.length * sizeof(float); 
    hipMemcpy(Vdevice.e, Vhost.e, size, hipMemcpyHostToDevice); 
} 
 
// Copy a device vector to a host vector 
void CopyFromDeviceVector(Vector Vhost, const Vector Vdevice){ 
    int size = Vdevice.length * sizeof(float); 
    hipMemcpy(Vhost.e, Vdevice.e, size, hipMemcpyDeviceToHost); 
} 
 
// Check if there is a device supporting CUDA 
bool CheckDevice() 
{ 
#if __DEVICE_EMULATION__ 
    return true; 
#else 
    int deviceCount; 
    hipGetDeviceCount(&deviceCount); 
    if (deviceCount == 0) 
        return false; 
    int dev; 
    for (dev = 0; dev < deviceCount; ++dev) { 
        hipDeviceProp_t deviceProp; 
        hipGetDeviceProperties(&deviceProp, dev); 
        if (strncmp(deviceProp.name, "Device Emulation", 16)) 
            break; 
    } 
    if (dev == deviceCount) 
        return false; 
    else { 
        hipSetDevice(dev); 
        return true; 
    } 
#endif 
} 
 
// Compute L2-norm error 
double ComputeL2NormError(const Vector V, const VectorD Vref){ 
    double error = 0; 
    double ref = 0; 
    for (int i = 0; i < V.length; ++i) { 
        double diff = V.e[i] - Vref.e[i]; 
        error += diff * diff; 
        ref += Vref.e[i] * Vref.e[i]; 
    } 
    double normRef = sqrt(ref); 
    if (fabs(ref) < 1e-15) { 
        printf("ComputeL2NormError(): Reference L2-norm is zero.\n"); 
        return -1; 
    } 
    double normError = sqrt(error); 
    return normError / normRef; 
} 
 

